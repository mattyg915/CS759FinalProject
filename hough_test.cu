#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <cstring>
#include "image_headers/image_utils.h"
#include "image_headers/hough.cuh"

extern "C" {
#define STB_IMAGE_IMPLEMENTATION
#include "image_headers/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "image_headers/stbi_image_write.h"
}

void insert(int curr_count, int curr_r, int curr_theta, int* best_count, int* best_r, int* best_theta, int numlines) {
	for (int i = 0; i < numlines; i++) {
		if (curr_count > best_count[i]) {
			for (int j = numlines - 1; j > i; j--) {
				best_count[j] = best_count[j-1];
				best_r[j] = best_r[j-1];
				best_theta[j] = best_theta[j-1];
			}
			best_count[i] = curr_count;
			best_r[i] = curr_r;
			best_theta[i] = curr_theta;
			return;
		}
	}
}

int main(int argc, char* argv[])
{
	// Must have exactly 1 command line argument
	if (argc != 2)
	{
		std::cerr << "Usage: ./main numlines" << std::endl;
		exit(1);
	}

	// take command line input (number of lines to display)
	int numlines = atoi(argv[1]);

	// initialize an array with mostly black but a couple of white pixels
	int width = 200;
	int height = 200;
	auto* pixels = new uint8_t[width * height];
	for (int i = 0; i < width*height; i++) {
		pixels[i] = 0;
	}
	pixels[510] = 255;
	pixels[2125] = 255;
	pixels[12175] = 255;
	pixels[8025] = 255;
	pixels[5678] = 255;

	// make a copy of pixels on the device
	int *intpixels = new int[width * height];
	for (int i = 0; i < width*height; i++) {
		intpixels[i] = (int)pixels[i];
	}
	int *dpixels;
	hipMalloc((void**)&dpixels, width * height * sizeof(int));
	hipMemcpy(dpixels, intpixels, width * height * sizeof(int), hipMemcpyHostToDevice);


	stbi_write_jpg("hough_output.jpg", width, height, 1, pixels, 100);

	// run the hough test to find the equation of the best line
	int* best_r = new int[numlines];
	int* best_theta = new int[numlines];
	int* best_count = new int[numlines];
	int max_r = (int)sqrt(width*width +  height*height);

	// SELF: Update this section to declare a 2d array for the (r, theta) pairs and increment in parallel
	int *line_matrix, *dline_matrix;
	line_matrix = (int*)malloc(2 * max_r * 360 * sizeof(int));
	hipMalloc((void**)&dline_matrix, 2 * max_r * 360 * sizeof(int));

	// Populate line_matrix with zeros
	for (size_t i = 0; i < 2 * max_r * 360; i++) {
		line_matrix[i] = 0;
	}

	// Copy line_matrix to device
	hipMemcpy(dline_matrix, line_matrix, 2 * max_r * 360 * sizeof(int), hipMemcpyHostToDevice);

	// Call kernel to accumulate counts in dline_matrix
	hough(dline_matrix, dpixels, width, height, max_r, 1024);

	// Copy line_matrix back to host
	hipMemcpy(line_matrix, dline_matrix, 2 * max_r * 360 * sizeof(int), hipMemcpyDeviceToHost);


	// Use updated line_matrix to compute best lines
	for (int r = -1 * max_r; r < max_r; r++) {
		for (int theta = 0; theta < 360; theta++) {
			int curr_count = line_matrix[360 * (r + max_r) + theta];
			if (curr_count > best_count[numlines - 1]) {
				insert(curr_count, r, theta, best_count, best_r, best_theta, numlines);
			}
		}
	}

	std::cout << best_r[0] << "\n";
	std::cout << best_theta[0] << "\n";
	std::cout << best_count[0] << "\n";

	// update pixels with best line drawn on it
	for (int k = 0; k < numlines; k++) {
		for (int i = 0; i < width; i++) {
			for (int j = 0; j < height; j++) {
				if (best_r[k] == (int)(i*cos(best_theta[k]) + j*sin(best_theta[k]))) {
					pixels[i * width + j] = 255;
				}
			}
		}
		// save image with each line added
		std::string s = "hough_output_with_" + std::to_string(k+1) + "_lines.jpg";
		int n = s.length();
		char s_char[n+1];
		strcpy(s_char, s.c_str());
		stbi_write_jpg(s_char, width, height, 1, pixels, 100);
	}

	//stbi_write_jpg("hough_output_with_lines.jpg", width, height, 1, pixels, 100);
	// free memory
	free(line_matrix);
	hipFree(dline_matrix);
	hipFree(dpixels);

	return 0;
}