#include "hip/hip_runtime.h"
#include "../image_headers/convolution.cuh"

float calcFx(const unsigned char* image, int i, int j, int width, int height) {
    if (0 <= i && i < width && 0 <= j && j < height)
    {
        return image[j * width + i];
    }
    else if ((0 <= i && i < width) || (0 <= j && j < height))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__global__ void convolve_kernel(unsigned char* image, unsigned char* output, int width, int height, const float *mask, int m)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            output[y * width + x] = 0;
            for (int i = 0; i < m; i++)
            {
                for (int j = 0; j < m; j++)
                {
                    float result = calcFx(image, x + i - m / 2, y + j - m / 2, width, height);
                    output[y * width + x] += mask[i * m + j] * result;
                }
            }
        }
    }
}