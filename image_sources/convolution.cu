#include "hip/hip_runtime.h"
#include "../image_headers/convolution.cuh"
#include <iostream>

__device__ float calcFx(const unsigned char* image, int i, int j, int width, int height) {
    if (0 <= i && i < width && 0 <= j && j < height)
    {
        printf("hit69\n");
        return image[j * width + i];
        printf("420\n");
    }
    else if ((0 <= i && i < width) || (0 <= j && j < height))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__global__ void convolve_kernel(unsigned char* image, unsigned char* output, int width, int height, const float *mask, int m)
{
    int x = threadIdx.x;
    int y = blockIdx.x;

    int output_index = blockIdx.x * blockDim.x + threadIdx.x;
    output[output_index] = 0;
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < m; j++)
        {
            float result = calcFx(image, x + i - m / 2, y + j - m / 2, width, height);
            printf("result is %f\n", result);
            output[output_index] += mask[i * m + j] * result;
        }
    }
}

void convolve(unsigned char* image, unsigned char* output, int width, int height, const float *mask, int m)
{
    int num_threads = 32;
    int num_blocks = (width * height - 1) / num_threads + 1;
    convolve_kernel<<<num_blocks, num_threads>>>(image, output, width, height, mask, m);
}