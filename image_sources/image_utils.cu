#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <cstdlib>
#include "../image_headers/stb_image.h"
#include "../image_headers/image_utils.cuh"

bool load_image(std::vector<unsigned char>& image, const char* filename, int& width, int& height, int& features, int force_features)
{
    // ... force_features = # 8-bit components per pixel ...
    // ... 'features' will always be the number that it would have been if you set force_features to 0
    unsigned char* data = stbi_load(filename, &width, &height, &features, force_features);
    if (data != nullptr)
    {
        image = std::vector<unsigned char>(data, data + width * height * force_features);
    }
    stbi_image_free(data);
    return (data != nullptr);
}

__global__ void rgb_to_greyscale_kernel(unsigned char* orig_image, unsigned char* output)
{
    int num_channels = 3;

    int output_index = blockIdx.x * blockDim.x + threadIdx.x;
    int index = num_channels * (output_index);

    uchar4 rgb = uchar4(orig_image + index);

    double grey = (0.299 * rgb.x) + (0.299 * rgb.w) + (0.299 * rgb.z);

    output[output_index] = grey;
}

/**
 * Takes an image with 3 channels, RGB, and converts to single channel greyscale
 * @param width width in elements of the image array
 * @param height height in elements of the image array
 * @param orig_image original image array
 * @param output array to output to
 */
void rgb_to_greyscale(int width, int height, std::vector<unsigned char>& image, unsigned char* output)
{
    int num_channels = 3;
    int input_size = width * height * num_channels;
    int output_size = width * height;
    int threads_per_block = 256;

    int num_blocks = (output_size - 1) / threads_per_block + 1;

    // copy data to the device
    unsigned char *dImage, *dOutput;
    hipMalloc((void **)&dImage, input_size * sizeof(unsigned char));
    hipMalloc((void **)&dOutput, output_size * sizeof(unsigned char));
    hipMemcpy(dImage, &image[0], input_size * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(dOutput, output, output_size * sizeof(unsigned char), hipMemcpyHostToDevice);

    // event timers
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    rgb_to_greyscale_kernel<<<num_blocks, threads_per_block>>>(dImage, dOutput);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float numMs;
    hipEventElapsedTime(&numMs, start, stop);

    std::cout << "to greyscale in cuda took " << numMs << "ms" << std::endl;

    // copy back
    hipMemcpy(output, dOutput, output_size * sizeof(unsigned char), hipMemcpyDeviceToHost);
}