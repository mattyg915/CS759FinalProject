#include <iostream>
#include <vector>
#include <chrono>
#include "image_headers/image_utils.h"
#include "image_headers/convolution.h"
#include "image_headers/canny.cuh"

extern "C" {
#define STB_IMAGE_IMPLEMENTATION
#include "image_headers/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "image_headers/stbi_image_write.h"
}

int main(int argc, char* argv[])
{
    #define CHANNEL_NUM 3

    using std::cout;
    using std::chrono::high_resolution_clock;
    using std::chrono::duration;

    // Must have exactly 1 command line argument
    if (argc != 2)
    {
        std::cerr << "Usage: ./main filename" << std::endl;
        exit(1);
    }

    char* filename = argv[1];

    int width, height, features;
    std::vector<unsigned char> image;
    bool image_loaded = load_image(image, filename, width, height, features, CHANNEL_NUM);

    if (!image_loaded)
    {
        std::cout << "Error loading image\n";
        exit(1);
    }

    cout << "Image width = " << width << std::endl;
    cout << "Image height = " << height << std::endl;

    auto* pixels = new unsigned char[width * height];
    auto* canny_output = new unsigned char[width * height];
    float* I_x = new float[width * height];
    float* I_y = new float[width * height];
    float* gradient = new float[width * height];
    float* theta = new float[width * height];

    // copy data to the device
    unsigned char *dpixels, *dcanny_output;
    float *dI_x, *dI_y;
    float *dgradient, *dtheta;

    size_t size  = width * height;

    hipMalloc((void **)&dpixels, size * sizeof(unsigned char));
    hipMalloc((void **)&dcanny_output, size * sizeof(unsigned char));
    
    hipMalloc((void **)&dI_x, size * sizeof(float));
    hipMalloc((void **)&dI_y, size * sizeof(float));
    
    hipMalloc((void **)&dgradient, size * sizeof(float));
    hipMalloc((void **)&dtheta, size * sizeof(float));

    hipMemcpy(dpixels, pixels, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(dcanny_output, canny_output, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(dI_x, I_x, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dI_y, I_y, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dgradient, gradient, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dtheta, theta, size * sizeof(float), hipMemcpyHostToDevice);

    rgb_to_greyscale(width, height, image, pixels);

    // generate timing variables
	hipEvent_t startEvent, stopEvent;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	// timing
	hipEventRecord(startEvent, 0);

	canny(pixels, canny_output, theta, gradient, I_x, I_y, width, height);

	// timing
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	std::cout << elapsedTime << "\n";

    // copy back
    hipMemcpy(canny_output, dcanny_output, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

    stbi_write_jpg("canny.jpg", width, height, 1, canny_output, 100);

    // Free device global memory
    hipFree(dpixels);  hipFree(dcanny_output);  hipFree(dI_x);

    // Free device global memory
    hipFree(dI_y);  hipFree(dgradient);  hipFree(dtheta);

    return 0;
}